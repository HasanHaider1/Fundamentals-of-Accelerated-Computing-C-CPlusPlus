
#include <hip/hip_runtime.h>
#include <stdio.h>

/*
 * Host function to initialize vector elements. This function
 * simply initializes each element to equal its index in the
 * vector.
 */

__global__
void initWith(float num, float *a, int N)
{
  int index = threadIdx.x + blockIdx.x * blockDim.x;
  if(index < N)
  {
    a[index] = num;
  }
}

/*
 * Device kernel stores into `result` the sum of each
 * same-indexed value of `a` and `b`.
 */

__global__
void addVectorsInto(float *result, float *a, float *b, int N)
{
  int index = threadIdx.x + blockIdx.x * blockDim.x;
  //int stride = blockDim.x * gridDim.x;
  
  if(index < N)
  {
  result[index] = a[index] +b[index];
  }

/*
 *for(int i = index; i < N; i += stride)
 *{
 *  result[i] = a[i] + b[i];
 *}
 */
}

/*
 * Host function to confirm values in `vector`. This function
 * assumes all values are the same `target` value.
 */

void checkElementsAre(float target, float *vector, int N)
{
  for(int i = 0; i < N; i++)
  {
    if(vector[i] != target)
    {
      printf("FAIL: vector[%d] - %0.0f does not equal %0.0f\n", i, vector[i], target);
      exit(1);
    }
  }
  printf("Success! All values calculated correctly.\n");
}

int main()
{
  const int N = 2<<24;
  size_t size = N * sizeof(float);

  float *a;
  float *b;
  float *c;

  hipMallocManaged(&a, size);
  hipMallocManaged(&b, size);
  hipMallocManaged(&c, size);

  size_t threadsPerBlock;
  size_t numberOfBlocks;

  /*
   * nsys should register performance changes when execution configuration
   * is updated.
   */
  
  int deviceId;
  hipGetDevice(&deviceId);
  
  hipDeviceProp_t props;
  hipGetDeviceProperties(&props, deviceId);
  
  threadsPerBlock = 256;
  
  //Calculating the number of Blocks needed
  int BlockNum = (N + threadsPerBlock - 1) / threadsPerBlock;
  
  //Calculating the closest multiple of the number of streaming processers
  numberOfBlocks = (((BlockNum - 1) / props.multiProcessorCount) + 1) * props.multiProcessorCount;
    
  
  hipError_t addVectorsErr;
  hipError_t asyncErr;
  
  //Changed the initWith function to be a cuda kernal.
  //The result is that there are now only 768 memory operations on the host only.
  //before this it was 11617 on device and 768 on host.
  //the add vectors kernal has ben sped up considerably as it now only takes 488817 nanoseconds.
  //the initwith Kernal takes 63004453 nanoseconds. 
  
  initWith<<<numberOfBlocks, threadsPerBlock>>>(3, a, N);
  initWith<<<numberOfBlocks, threadsPerBlock>>>(4, b, N);
  initWith<<<numberOfBlocks, threadsPerBlock>>>(0, c, N);
  
  asyncErr = hipDeviceSynchronize();
  if(asyncErr != hipSuccess) printf("Error: %s\n", hipGetErrorString(asyncErr));

  addVectorsInto<<<numberOfBlocks, threadsPerBlock>>>(c, a, b, N);

  addVectorsErr = hipGetLastError();
  if(addVectorsErr != hipSuccess) printf("Error: %s\n", hipGetErrorString(addVectorsErr));

  asyncErr = hipDeviceSynchronize();
  if(asyncErr != hipSuccess) printf("Error: %s\n", hipGetErrorString(asyncErr));

  checkElementsAre(7, c, N);

  hipFree(a);
  hipFree(b);
  hipFree(c);
}
